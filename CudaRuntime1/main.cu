#include "hip/hip_runtime.h"
﻿#include <cstdlib>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel1()
{
    printf("Hello, world!\n");
}

__global__ void kernel2()
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    printf("I am from %d block, %d thread (global index: %d)\n", blockIdx.x, threadIdx.x, k);
}

__global__ void kernel3(int* a, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        a[i] += i;
    }
}

int main()
{
    int n = 10;
    const int block_size = 256;
    const int num_block = (n + block_size - 1) / block_size;

    //task1
    kernel1 << <num_block, block_size >> > ();

    //task2
    kernel2 << <num_block, block_size >> > ();

    //task3
    int* a = new int[n];
    int* a_gpu;
    for (int i = 0; i < n; i++) {
        a[i] = 0;
    }
    hipMalloc((void**)&a_gpu, n * sizeof(int));
    hipMemcpy(a_gpu, a, n * sizeof(int), hipMemcpyHostToDevice);
    kernel3 << <num_block, block_size >> > (a_gpu, n);
    hipMemcpy(a, a_gpu, n * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        printf("%d\n", a[i]);
    }
    delete[] a;
    hipFree(a_gpu);

    hipDeviceSynchronize();
    return 0;
}